#include "hip/hip_runtime.h"
#include "vecadd.h"
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
	 std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
      if (abort) exit(code);
   }
}


__global__ void kernel_vecadd(int* vec1, int* vec2, int* result, int N)
{
  int idx = blockIdx.x;
  result[idx] = vec1[idx] + vec2[idx];
}


void gpu_vecadd(int* vec1, int* vec2, int* result, int N)
{

  int* d_vec1, *d_vec2, *d_result;
  hipMalloc((void**) &d_vec1, sizeof(int) * N);
  hipMalloc((void**) &d_vec2, sizeof(int) * N);
  hipMalloc((void**) &d_result, sizeof(int) * N);

  hipMemcpy(d_vec1, vec1, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_vec2, vec2, sizeof(int) * N, hipMemcpyHostToDevice);
  
  kernel_vecadd<<<N, 1>>>(d_vec1, d_vec2, d_result, N);

  hipMemcpy(result, d_result, sizeof(int) * N, hipMemcpyDeviceToHost);

  hipFree(d_vec1);
  hipFree(d_vec2);
  hipFree(d_result);
}
