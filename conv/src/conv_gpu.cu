#include "hip/hip_runtime.h"
#include "conv.h"
#include "conv_common.h"
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
	 std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
      if (abort) exit(code);
   }
}


__global__ void simple_conv(float* in_data, float* out_data, uint width, uint height, float* mask, uint mask_width, uint mask_height)
{
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx_x >= width || idx_y >= height)
	return;
  
  int mask_hw = mask_width/2;
  int mask_hh = mask_height/2;

  float sum = 0.0;
  float mask_sum = 0.0;
  for (uint x = 0 ; x < mask_width ; ++x)
  {
	for (uint y = 0 ; y < mask_height ; ++y)
	{
	  int px = idx_x + (x - mask_hw);
	  int py = idx_y + (y - mask_hh);

	  if (px < 0 || px >= width || py < 0 || py >= height)
		continue;

	  float m_value = mask[x + y * mask_width];
	  sum += m_value * in_data[px + py * width];
	  mask_sum += m_value;
	}
  }
  out_data[idx_x + idx_y * width] = sum / mask_sum;
}

float* gpu_conv(float* image, uint width, uint height, float* mask, uint mask_width, uint mask_height, float& exec_time)
{
  uint size = width * height;
  float* out_image = new float[size];
  
  uint mask_size = mask_width * mask_height;
  
  float *d_in, *d_out, *d_mask;
  gpuErrchk( hipMalloc((void**) &d_in, size * sizeof(float)) );
  gpuErrchk( hipMalloc((void**) &d_out, size * sizeof(float)) );
  gpuErrchk( hipMalloc((void**) &d_mask, mask_size * sizeof(float)) );

  gpuErrchk( hipMemcpy(d_in, image, size * sizeof(float), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(d_mask, mask, mask_size * sizeof(float), hipMemcpyHostToDevice) );

  // Determining best threads per block & block number to run the kernel
  struct hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
  float dim = sqrt((float)maxThreadsPerBlock);
  const dim3 blockDim(dim, dim);
  const dim3 numBlocks(width/dim, height/dim);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  simple_conv<<<numBlocks, blockDim>>>(d_in, d_out, width, height,
									   d_mask, mask_width, mask_height);
  hipEventRecord(stop);

  gpuErrchk( hipMemcpy(out_image, d_out, size * sizeof(float), hipMemcpyDeviceToHost) );

  hipEventSynchronize(stop);
  hipEventElapsedTime(&exec_time, start, stop);
  
  gpuErrchk( hipFree(d_in) );
  gpuErrchk( hipFree(d_out) );
  gpuErrchk( hipFree(d_mask) );

  return out_image;
}
